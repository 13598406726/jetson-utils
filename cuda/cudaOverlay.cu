#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaOverlay.h"


template<typename T>
__global__ void gpuRectFill( T* input, T* output, int width, int height,
					    float4* rects, int numRects, float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	T px = input[ y * width + x ];

	const float fx = x;
	const float fy = y;
	
	const float alpha = color.w / 255.0f;
	const float ialph = 1.0f - alpha;
	
	for( int nr=0; nr < numRects; nr++ )
	{
		const float4 r = rects[nr];
	
		if( fy >= r.y && fy <= r.w && fx >= r.x && fx <= r.z )
		{
			px.x = alpha * color.x + ialph * px.x;
			px.y = alpha * color.y + ialph * px.y;
			px.z = alpha * color.z + ialph * px.z;
		}
	}
	
	output[y * width + x] = px;	 
}

template<typename T>
__global__ void gpuRectFillBox( T* input, T* output, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight )
		return;

	T px = input[ y * imgWidth + x ];

	const float alpha = color.w / 255.0f;
	const float ialph = 1.0f - alpha;

	px.x = alpha * color.x + ialph * px.x;
	px.y = alpha * color.y + ialph * px.y;
	px.z = alpha * color.z + ialph * px.z;
	
	output[y * imgWidth + x] = px;
}

template<typename T>
hipError_t launchRectFill( T* input, T* output, uint32_t width, uint32_t height, float4* rects, int numRects, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || !rects || numRects == 0 )
		return hipErrorInvalidValue;

	// if input and output are the same image, then we can use the faster method
	// which draws 1 box per kernel, but doesn't copy pixels that aren't inside boxes
	if( input == output )
	{
		for( int n=0; n < numRects; n++ )
		{
			const int boxWidth = (int)(rects[n].z - rects[n].x);
			const int boxHeight = (int)(rects[n].w - rects[n].y);

			// launch kernel
			const dim3 blockDim(8, 8);
			const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));

			gpuRectFillBox<T><<<gridDim, blockDim>>>(input, output, width, height, (int)rects[n].x, (int)rects[n].y, boxWidth, boxHeight, color); 
		}
	}
	else
	{
		// launch kernel
		const dim3 blockDim(8, 8);
		const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

		gpuRectFill<T><<<gridDim, blockDim>>>(input, output, width, height, rects, numRects, color);
	}

	return hipGetLastError();
}

// cudaRectFill
hipError_t cudaRectFill( void* input, void* output, imageFormat format, uint32_t width, uint32_t height, float4* rects, int numRects, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || !rects || numRects == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	if( format == IMAGE_RGB8 )
		return launchRectFill<uchar3>((uchar3*)input, (uchar3*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGBA8 )
		return launchRectFill<uchar4>((uchar4*)input, (uchar4*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGB32F )
		return launchRectFill<float3>((float3*)input, (float3*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGBA32F )
		return launchRectFill<float4>((float4*)input, (float4*)output, width, height, rects, numRects, color); 
	else
		return hipErrorInvalidValue;
}
